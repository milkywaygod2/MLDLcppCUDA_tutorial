#include "hip/hip_runtime.h"
#include "_pch.h"

#define BLOCK_SIZE 32

//--- mat_slice_row : Ư�� ��~���� ����
__global__ void mat_slice_rows_kernel(const float* __restrict__ _srcArr1D, float* __restrict__ _dstArr1D, int _srcArrCols, int _srcArrRows, int _startRow, int _offsetRow) {
    int threadPos_row = blockIdx.y * blockDim.y + threadIdx.y;
    int threadPos_col = blockIdx.x * blockDim.x + threadIdx.x; //printf("%d x %d\n", row, col);
    if(_startRow <= threadPos_row && threadPos_row < _startRow + _offsetRow 
        && threadPos_row < _srcArrRows && threadPos_col < _srcArrCols) { 
        _dstArr1D[threadPos_col * _offsetRow + (threadPos_row - _startRow)] = _srcArr1D[threadPos_col * _srcArrRows + threadPos_row];
    //printf("startRow:%d offsetRow:%d %dx%d [%d] %f\n", 
    // _startRow, _offsetRow, threadPos_row, threadPos_col, threadPos_col * _srcArrRows + threadPos_row, _srcArr1D[threadPos_col * _srcArrRows + threadPos_row]);
    }
}
void mat_slice_rows_kernel_exec(const float* _srcArr1D, float* _dstArr1D, int _srcArrCols, int _srcArrRows, int _startRow, int _offsetRow) {
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((_srcArrCols + block.x - 1) / block.x, (_srcArrRows + block.y - 1) / block.y);
    //printf("srcArrCols:%d srcArrRows:%d startRow:%d offsetRow:%d\n", 
    // srcArrCols, srcArrRows, _startRow, _offsetRow);

    /* lunch kernel */
    mat_slice_rows_kernel KERNEL_ARG2(grid, block) (_srcArr1D, _dstArr1D, _srcArrCols, _srcArrRows, _startRow, _offsetRow);
    hipDeviceSynchronize();
}

//--- mat_join_row : Ư�� ��~�࿡ ����
__global__ void mat_join_rows_kernel(const float* __restrict__ _srcArr1D, float* __restrict__ _dstArr1D, int _srcArrCols, int _srcArrRows, int _startRow, int _offsetRow) {
    int threadPos_row = blockIdx.y * blockDim.y + threadIdx.y;
    int threadPos_col = blockIdx.x * blockDim.x + threadIdx.x; //printf("%d x %d\n", row, col);
    if(_startRow <= threadPos_row && threadPos_row < _startRow + _offsetRow
        && threadPos_row < _srcArrCols && threadPos_col < _srcArrCols) {
        _dstArr1D[threadPos_col * _srcArrCols + threadPos_row] = _srcArr1D[threadPos_col * _offsetRow + (threadPos_row - _startRow)];
    //printf("startRow:%d offsetRow:%d %dx%d [%d] %f\n", 
    // _startRow, _offsetRow, threadPos_row, threadPos_col, threadPos_col * _srcArrRows + threadPos_row, _srcArr1D[threadPos_col * _srcArrRows + threadPos_row]);
    }
}
void mat_join_rows_kernel_exec(const float* _srcArr1D, float* _dstArr1D, int _srcArrCols, int _srcArrRows, int _startRow, int _offsetRow) {
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((_srcArrCols + block.x - 1) / block.x, (_srcArrRows + block.y - 1) / block.y);
    //printf("srcArrCols:%d srcArrRows:%d startRow:%d offsetRow:%d\n", 
    // srcArrCols, srcArrRows, _startRow, _offsetRow);

    /* lunch kernel */
    mat_join_rows_kernel KERNEL_ARG2(grid, block) (_srcArr1D, _dstArr1D, _srcArrCols, _srcArrRows, _startRow, _offsetRow);
    hipDeviceSynchronize();
}

//--- mat_fill_1
__global__ void mat_fill_1_kernel(const float* __restrict__ _srcArr1D, float* __restrict__ _dstArr1D, int _srcArrCols, int _srcArrRows) { //__device__ȣ����� �ʿ���� �ܼ��� ���� (�׷��� for�����ٴ� �ܼ�Ŀ���̶� ������)
    int threadPos_row = blockIdx.y * blockDim.y + threadIdx.y;
    int threadPos_col = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadPos_row < _srcArrCols && threadPos_col < _srcArrRows) { _dstArr1D[threadPos_row * _srcArrRows + threadPos_col] = 1.0; }
}
void mat_fill_1_kernel_exec(const float* _srcArr1D, float* _dstArr1D, int _srcArrCols, int _srcArrRows) {
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((_srcArrRows + block.x - 1) / block.x, (_srcArrCols + block.y - 1) / block.y);

    /* lunch kernel */
    mat_fill_1_kernel KERNEL_ARG2(grid, block) (_srcArr1D, _dstArr1D, _srcArrCols, _srcArrRows);
    hipDeviceSynchronize();
}

//--- mat_sqrt_withplus_k
__device__ __forceinline__ float mat_sqrt_withplus_k(float _srcEachArr1D, float _k) { return std::sqrt(_srcEachArr1D + _k); }
__global__ void mat_sqrt_withplus_k_kernel(const float* __restrict__ _srcArr1D, float* __restrict__ _dstArr1D, int _srcArrCols, int _srcArrRows, float _k) {
    int threadPos_row = blockIdx.y * blockDim.y + threadIdx.y; //��ü �׸��峻 ��Ϲ�ȣ * �������ȣ + �����峻 �ε���
    int threadPos_col = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadPos_row < _srcArrCols && threadPos_col < _srcArrRows) {
        _dstArr1D[threadPos_row * _srcArrRows + threadPos_col] = mat_sqrt_withplus_k(_srcArr1D[threadPos_row * _srcArrRows + threadPos_col], _k); }
}
void mat_sqrt_withplus_k_kernel_exec(const float* _srcArr1D, float* _dstArr1D, int _srcArrCols, int _srcArrRows, float _k) {
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((_srcArrRows + block.x - 1) / block.x, (_srcArrCols + block.y - 1) / block.y);

    /* lunch kernel */
    mat_sqrt_withplus_k_kernel KERNEL_ARG2(grid, block) (_srcArr1D, _dstArr1D, _srcArrCols, _srcArrRows, _k);
    hipDeviceSynchronize();
}

//--- mat_sqrt_withplus_k_d
__device__ __forceinline__ float mat_sqrt_withplus_k_d(float _srcEachArr1D, float _k) { return 0.5 * 1.0 / std::sqrt(_srcEachArr1D + _k); } /*return 0.5 * pow(a + alpha, -0.5f);*/
__global__ void mat_sqrt_withplus_k_d_kernel(const float* __restrict__ _srcArr1D, float* __restrict__ _dstArr1D, int _srcArrCols, int _srcArrRows, float _k) {
    int threadPos_row = blockIdx.y * blockDim.y + threadIdx.y; //��ü �׸��峻 ��Ϲ�ȣ * �������ȣ + �����峻 �ε���
    int threadPos_col = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadPos_row < _srcArrCols && threadPos_col < _srcArrRows) { 
        _dstArr1D[threadPos_row * _srcArrRows + threadPos_col] = mat_sqrt_withplus_k_d(_srcArr1D[threadPos_row * _srcArrRows + threadPos_col], _k); }
}
void mat_sqrt_withplus_k_d_kernel_exec(const float* _srcArr1D, float* _dstArr1D, int _srcArrCols, int _srcArrRows, float _k) {
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((_srcArrRows + block.x - 1) / block.x, (_srcArrCols + block.y - 1) / block.y);

    /* lunch kernel */
    mat_sqrt_withplus_k_d_kernel KERNEL_ARG2(grid, block) (_srcArr1D, _dstArr1D, _srcArrCols, _srcArrRows, _k);
    hipDeviceSynchronize();
}

//--- mat_mat_log_withplus_k
__device__ __forceinline__ float mat_log_withplus_k(float _srcEachArr1D, float _k) { return std::log(_srcEachArr1D + _k); }
__global__ void mat_log_withplus_k_kernel(const float* __restrict__ _srcArr1D, float* __restrict__ _dstArr1D, int _srcArrCols, int _srcArrRows, float _k) {
    int threadPos_row = blockIdx.y * blockDim.y + threadIdx.y; //��ü �׸��峻 ��Ϲ�ȣ * �������ȣ + �����峻 �ε���
    int threadPos_col = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadPos_row < _srcArrCols && threadPos_col < _srcArrRows) { 
        _dstArr1D[threadPos_row * _srcArrRows + threadPos_col] = mat_log_withplus_k(_srcArr1D[threadPos_row * _srcArrRows + threadPos_col], _k); }
}
void mat_log_withplus_k_kernel_exec(const float* _srcArr1D, float* _dstArr1D, int _srcArrCols, int _srcArrRows, float _k) {
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((_srcArrRows + block.x - 1) / block.x, (_srcArrCols + block.y - 1) / block.y);

    /* lunch kernel */
    mat_log_withplus_k_kernel KERNEL_ARG2(grid, block) (_srcArr1D, _dstArr1D, _srcArrCols, _srcArrRows, _k);
    hipDeviceSynchronize();
}

//--- mat_A_mul_B
__global__ void mat_A_mul_B_kernel(const float* __restrict__ _src1Arr1D, const float* __restrict__ _src2Arr1D,  float* __restrict__ _dstArr1D, int _srcArrCols, int _srcArrRows) {
    int threadPos_row = blockIdx.y * blockDim.y + threadIdx.y;
    int threadPos_col = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadPos_row < _srcArrCols && threadPos_col < _srcArrRows) { 
        _dstArr1D[threadPos_row * _srcArrRows + threadPos_col] = _src1Arr1D[threadPos_row * _srcArrRows + threadPos_col] * _src2Arr1D[threadPos_row * _srcArrRows + threadPos_col]; }
}
void mat_A_mul_B_kernel_exec(const float* _src1Arr1D, const float* _src2Arr1D,  float* _dstArr1D, int _srcArrCols, int _srcArrRows) {
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((_srcArrRows + block.x - 1) / block.x, (_srcArrCols + block.y - 1) / block.y);

    /* lunch kernel */
    mat_A_mul_B_kernel KERNEL_ARG2(grid, block) (_src1Arr1D, _src2Arr1D, _dstArr1D, _srcArrCols, _srcArrRows);
    hipDeviceSynchronize();
}

//--- mat_aA_mul_bB_plusEqual
__global__ void mat_aA_mul_bB_plusEqual_kernel( const float* __restrict__ _src1Arr1D, const float* __restrict__ _src2Arr1D, float* __restrict__ _dstArr1D, float _a, float _b, int _srcArrCols, int _srcArrRows) {
    int threadPos_row = blockIdx.y * blockDim.y + threadIdx.y;
    int threadPos_col = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadPos_row < _srcArrCols && threadPos_col < _srcArrRows) { 
        _dstArr1D[threadPos_row * _srcArrRows + threadPos_col] += _a * _src1Arr1D[threadPos_row * _srcArrRows + threadPos_col] * _b * _src2Arr1D[threadPos_row * _srcArrRows + threadPos_col]; }
}
void mat_aA_mul_bB_plusEqual_kernel_exec(const float* _src1Arr1D, const float* _src2Arr1D, float* _dstArr1D, float _a, float _b, int _srcArrCols, int _srcArrRows) {
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((_srcArrRows + block.x - 1) / block.x, (_srcArrCols + block.y - 1) / block.y);

    /* lunch kernel */
    mat_aA_mul_bB_plusEqual_kernel KERNEL_ARG2(grid, block) (_src1Arr1D, _src2Arr1D, _dstArr1D, _a, _b, _srcArrCols, _srcArrRows);
    hipDeviceSynchronize();
}

//--- mat_A_div_B
__global__ void mat_A_div_B_kernel(const float* __restrict__ _src1Arr1D, const float* __restrict__ _src2Arr1D, float* __restrict__ _dstArr1D, int _srcArrCols, int _srcArrRows) {
    int threadPos_row = blockIdx.y * blockDim.y + threadIdx.y;
    int threadPos_col = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadPos_row < _srcArrCols && threadPos_col < _srcArrRows) { 
        _dstArr1D[threadPos_row * _srcArrRows + threadPos_col] = _src1Arr1D[threadPos_row * _srcArrRows + threadPos_col] / _src2Arr1D[threadPos_row * _srcArrRows + threadPos_col]; }
}
void mat_A_div_B_kernel_exec(const float* _src1Arr1D, const float* _src2Arr1D, float* _dstArr1D, int _srcArrCols, int _srcArrRows) {
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((_srcArrRows + block.x - 1) / block.x, (_srcArrCols + block.y - 1) / block.y);

    /* lunch kernel */
    mat_A_div_B_kernel KERNEL_ARG2(grid, block) (_src1Arr1D, _src2Arr1D, _dstArr1D, _srcArrCols, _srcArrRows);
    hipDeviceSynchronize();
}


